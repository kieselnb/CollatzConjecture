#include "hip/hip_runtime.h"
/**
 * @file collatz_runner_gpu.cpp
 *
 * This file contains the definition of the CollatzRunnerGPU class
 */

#include <thread>
#include <iostream>

#include <hip/hip_runtime.h>

#include "collatz_runner_gpu.cuh"
#include "collatz_counter.hpp"

__global__
void collatz(uint64_t start, int stride, int *status) {
    int k = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (k < stride) {
        uint64_t myNum = start + k;

        while (myNum > 1) {
            if (myNum & 2 == 0) {
                myNum = myNum >> 1;
            }
            else {
                myNum = ((myNum * 3) + 1) >> 1;
            }
        }

        if (myNum != 1) {
            *status = 0;
        }
    }
}

using namespace std;

CollatzRunnerGPU::CollatzRunnerGPU(CollatzCounter &counter)
    : CollatzRunner(counter)
{

}

void CollatzRunnerGPU::start() {
    collatzThread = new thread(runner, ref(*this));
}

void CollatzRunnerGPU::join() {
    collatzThread->join();
}

void CollatzRunnerGPU::runner(CollatzRunnerGPU& self) {
    self._stride = 1 << 21;

    int status, *d_status;
    hipError_t err = hipMalloc(&d_status, sizeof(int));
    if (err != hipSuccess) {
        cout << "hipMalloc failed, did you forget optirun?" << endl;
        return;
    }

    while (true) {
        status = 1;
        hipMemcpy(d_status, &status, sizeof(int), hipMemcpyHostToDevice);

        uint64_t start = self._counter.take(self._stride);
        collatz<<<(self._stride+255)/256, 256>>>(start, self._stride, d_status);

        hipMemcpy(&status, d_status, sizeof(int), hipMemcpyDeviceToHost);
        if (status == 0) {
            cout << "WE BROKE SOMETHING" << endl;
        }
    }
}
